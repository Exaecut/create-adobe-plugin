
#include <hip/hip_runtime.h>
__global__ void colorTransitionKernel(float4* outImage, int width, int height, float time) {
    // Calculate the thread’s x and y position in the grid
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within image bounds
    if (x >= width || y >= height) return;

    // Calculate the pixel index
    int pixelIndex = y * width + x;

    // Calculate intensity using sin(time)
    float intensity = 0.5f + 0.5f * sinf(time);

    // Set the pixel color (black to white transition)
    outImage[pixelIndex] = make_float4(intensity, intensity, intensity, 1.0f);
}